
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <math.h>
// #include <chrono>

struct point {
    double x;
    double y;
    double z;

    __host__ __device__ point() {}
    __host__ __device__ point(double x, double y, double z) : x(x), y(y), z(z) {}
};

std::istream& operator>>(std::istream& in, point& p) {
    in >> p.x >> p.y >> p.z;
    return in;
}

std::ostream& operator<<(std::ostream& out, point& p) {
    out << p.x << " " <<  p.y << " " << p.z;
    return out;
}

__host__ __device__ point operator+(point a, point b) {
    return point(
        a.x + b.x,
        a.y + b.y,
        a.z + b.z
    );
}

__host__ __device__ point operator-(point a, point b) {
    return point(
        a.x - b.x,
        a.y - b.y,
        a.z - b.z
    );
}

__host__ __device__ point operator*(point a, double b) {
    return point(
        a.x * b,
        a.y * b,
        a.z * b
    );
}

__host__ __device__ double dot(point a, point b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ point prod(point a, point b) {
    return point(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__host__ __device__ point normalize(point v) {
    double l = sqrt(dot(v, v));
    return point(
        v.x / l,
        v.y / l,
        v.z / l
    );
}

__host__ __device__ point multiply(point a, point b, point c, point v) {
    return point(
        a.x * v.x + b.x * v.y + c.x * v.z,
        a.y * v.x + b.y * v.y + c.y * v.z,
        a.z * v.x + b.z * v.y + c.z * v.z
    );
}

struct polygon {
    point a;
    point b;
    point c;
    uchar4 color;

    __host__ __device__ polygon() {}
    __host__ __device__ polygon(point a, point b, point c, uchar4 color) : a(a), b(b), c(c), color(color) {}
};

__host__ __device__ uchar4 make_ray(
    point position, point direction, 
    point light_position, uchar4 light_color, 
    polygon* polygons, int polygons_amount
) {
    int i_min = -1;
    double ts_min;

    for (int i = 0; i < polygons_amount; ++i) {
        point e1 = polygons[i].b - polygons[i].a;
        point e2 = polygons[i].c - polygons[i].a;
        point p = prod(direction, e2);
        double div = dot(p, e1);
        if (fabs(div) < 1e-10) {
            continue;
        }
        point t = position - polygons[i].a;
        double u = dot(p, t) / div;
        if (u < 0.0 || u > 1.0) {
            continue;
        }
        point q = prod(t, e1);
        double v = dot(q, direction) / div;
        if (v < 0.0 || v + u > 1.0) {
            continue;
        }
        double ts = dot(q, e2) / div; 
        if (ts < 0.0) {
            continue;
        }
        if (i_min == -1 || ts < ts_min) {
            i_min = i;
            ts_min = ts;
        }
    }

    if (i_min == -1) {
        return make_uchar4(0, 0, 0, 255);
    }

    point new_position = direction * ts_min + position;
    point new_direction = light_position - new_position;
    double length = sqrt(dot(new_direction, new_direction));
    new_direction = normalize(new_direction);

    for (int i = 0; i < polygons_amount; ++i) {
        point e1 = polygons[i].b - polygons[i].a;
        point e2 = polygons[i].c - polygons[i].a;
        point p = prod(new_direction, e2);
        double div = dot(p, e1);
        if (fabs(div) < 1e-10)
            continue;
        point t = new_position - polygons[i].a;
        double u = dot(p, t) / div;
        if (u < 0.0 || u > 1.0)
            continue;
        point q = prod(t, e1);
        double v = dot(q, new_direction) / div;
        if (v < 0.0 || v + u > 1.0)
            continue;
        double ts = dot(q, e2) / div; 
        if (ts > 0.0 && ts < length && i != i_min) {
            return make_uchar4(0, 0, 0, 255);
        }
    }

    return make_uchar4(
        polygons[i_min].color.x * light_color.x,
        polygons[i_min].color.y * light_color.y,
        polygons[i_min].color.z * light_color.z,
        255
    );
}

__host__ __device__ void cpu_render(
    uchar4* data, 
    point camera_position, point camera_view, 
    int width, int height, double view_angle, 
    point light_position, uchar4 light_color, 
    polygon* polygons, int polygons_amount
) {
    double dw = 2. / (width - 1.);
    double dh = 2. / (height - 1.);
    double z = 1. / tan(view_angle * M_PI / 360.);

    point bz = normalize(camera_view - camera_position);
    point bx = normalize(prod(bz, {0., 0., 1.}));
    point by = normalize(prod(bx, bz));

    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < height; ++j) {
            point v = point(-1. + dw * i, (-1. + dh * j) * height / width, z);
            point dir = multiply(bx, by, bz, v);
            data[(height - 1 - j) * width + i] = make_ray(camera_position, normalize(dir), light_position, light_color, polygons, polygons_amount);
        }
    }
}

__global__ void gpu_render(
    uchar4* data,
    point camera_pos, point camera_view, 
    int width, int height, double view_angle,  
    point light_position, uchar4 light_color, 
    polygon* polygons, int polygons_amount
) {
    int i_x = blockDim.x * blockIdx.x + threadIdx.x;
    int i_y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    double dw = 2. / (width - 1.);
    double dh = 2. / (height - 1.);
    double z = 1. / tan(view_angle * M_PI / 360.);

    point bz = normalize(camera_view - camera_pos);
    point bx = normalize(prod(bz, {0., 0., 1.}));
    point by = normalize(prod(bx, bz));

    for (int i = i_x; i < width; i += offset_x) {
        for (int j = i_y; j < height; j += offset_y) {
            point v = point(-1. + dw * i, (-1. + dh * j) * height / width, z);
            point dir = multiply(bx, by, bz, v);
            data[(height - 1 - j) * width + i] = make_ray(camera_pos, normalize(dir), light_position, light_color, polygons, polygons_amount);
        }
    }
}

__host__ __device__ void cpu_smoothing(uchar4* data, uchar4* smoothing_data, int width, int height, int sqrt_rpp) {
    for (int x = 0; x < width; ++x) {
        for (int y = 0; y < height; ++y) {
            uint4 temp = make_uint4(0, 0, 0, 0);
            for (int i = 0; i < sqrt_rpp; ++i) {
                for (int j = 0; j < sqrt_rpp; ++j) {
                    uchar4 cur_pixel = data[width * sqrt_rpp * (y * sqrt_rpp + j) + (x * sqrt_rpp + i)];
                    temp.x += cur_pixel.x;
                    temp.y += cur_pixel.y;
                    temp.z += cur_pixel.z;
                }
            }
            int rpp = sqrt_rpp * sqrt_rpp;
            smoothing_data[y * width + x] = make_uchar4(temp.x / rpp, temp.y / rpp, temp.z / rpp, 255);
        }
    }
}

__global__ void gpu_smoothing(uchar4* data, uchar4* smoothing_data, int width, int height, int sqrt_rpp) {
    int i_x = blockDim.x * blockIdx.x + threadIdx.x;
    int i_y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    for (int x = i_x; x < width; x += offset_x) {
        for (int y = i_y; y < height; y += offset_y) {
            uint4 temp = make_uint4(0, 0, 0, 0);
            for (int i = 0; i < sqrt_rpp; ++i) {
                for (int j = 0; j < sqrt_rpp; ++j) {
                    uchar4 cur_pixel = data[width * sqrt_rpp * (y * sqrt_rpp + j) + (x * sqrt_rpp + i)];
                    temp.x += cur_pixel.x;
                    temp.y += cur_pixel.y;
                    temp.z += cur_pixel.z;
                }
            }
            int rpp = sqrt_rpp * sqrt_rpp;
            smoothing_data[y * width + x] = make_uchar4(temp.x / rpp, temp.y / rpp, temp.z / rpp, 255);
        }
    }
}


struct frames_params {
    int amount;
    std::string path_to_save_frames;
    int width, height;
    double view_angle;
};

std::istream& operator>>(std::istream& in, frames_params& f) {
    in >> f.amount >> f.path_to_save_frames >> f.width >> f.height >> f.view_angle;
    return in;
}

struct camera_params {
    double r0c, z0c, phi0c, arc, azc, wrc, wzc, wphic, prc, pzc;
    double r0n, z0n, phi0n, arn, azn, wrn, wzn, wphin, prn, pzn;
};

std::istream& operator>>(std::istream& in, camera_params& c) {
    in >> c.r0c >> c.z0c >> c.phi0c >> c.arc >> c.azc >> c.wrc >> c.wzc >> c.wphic >> c.prc >> c.pzc;
    in >> c.r0n >> c.z0n >> c.phi0n >> c.arn >> c.azn >> c.wrn >> c.wzn >> c.wphin >> c.prn >> c.pzn;
    return in;
}

struct figure_params {
    point center;
    uchar4 color;
    double radius;
};

std::istream& operator>>(std::istream& in, figure_params& f) {
    in >> f.center;

    double r, g, b;
    in >> r >> g >> b;
    f.color = make_uchar4(r * 255, g * 255, b * 255, 255);

    in >> f.radius;
    return in;
}

struct floor_params {
    point p1, p2, p3, p4;
    uchar4 color;
};

std::istream& operator>>(std::istream& in, floor_params& f) {
    in >> f.p1 >> f.p2 >> f.p3 >> f.p4;

    double r, g, b;
    in >> r >> g >> b;
    f.color = make_uchar4(r * 255, g * 255, b * 255, 255);

    return in;
}

struct light_params {
    point position;
    uchar4 color;
    double sqrt_rpp;
};

std::istream& operator>>(std::istream& in, light_params& l) {
    in >> l.position;

    double r, g, b;
    in >> r >> g >> b;
    l.color = make_uchar4(r * 255, g * 255, b * 255, 255);

    in >> l.sqrt_rpp;
    return in;
}


class app {
    bool use_gpu;
    int polygons_amount = 62;
    int x_blocks_amount = 8, x_threads_amount = 8;
    int y_blocks_amount = 8, y_threads_amount = 8;

    frames_params frames;
    camera_params camera;
    figure_params tetrahedron;
    figure_params dodecahedron;
    figure_params icosahedron;
    floor_params floor;
    light_params light;

    void init_floor(polygon* polygons) {
        polygons[0] = polygon(floor.p1, floor.p2, floor.p3, floor.color);
        polygons[1] = polygon(floor.p1, floor.p3, floor.p4, floor.color);
    }

    void init_tetrahedron(polygon* polygons) {
        double a = 4. / sqrt(6) * tetrahedron.radius;

        point v1 = point(tetrahedron.center.x, tetrahedron.center.y + tetrahedron.radius, tetrahedron.center.z);
        point v2 = point(tetrahedron.center.x + 0.578 * a, tetrahedron.center.y - 1/3 * tetrahedron.radius, tetrahedron.center.z);
        point v3 = point(tetrahedron.center.x - 0.289 * a, tetrahedron.center.y - 1/3 * tetrahedron.radius, tetrahedron.center.z + 0.5 * a);
        point v4 = point(tetrahedron.center.x - 0.289 * a, tetrahedron.center.y - 1/3 * tetrahedron.radius, tetrahedron.center.z - 0.5 * a);
    
        polygons[2] = polygon(v1, v2, v3, tetrahedron.color);
        polygons[3] = polygon(v1, v3, v4, tetrahedron.color);
        polygons[4] = polygon(v1, v2, v4, tetrahedron.color);
        polygons[5] = polygon(v2, v3, v4, tetrahedron.color);
    }

    void init_dodecahedron(polygon* polygons) {
        double a = (1. + sqrt(5.)) / 2.;
        double b = 1. / a;

        point v[] = {
            point(-b, 0., a),
            point(b, 0., a),
            point(-1., 1., 1.),
            point(1., 1., 1.),
            point(1., -1., 1.),
            point(-1., -1., 1.),
            point(0., -a, b),
            point(0., a, b),
            point(-a, -b, 0.),
            point(-a, b, 0.),
            point(a, b, 0.),
            point(a, -b, 0.),
            point(0., -a, -b),
            point(0., a, -b),
            point(1., 1., -1.),
            point(1., -1., -1.),
            point(-1., -1., -1.),
            point(-1., 1., -1.),
            point(b, 0., -a),
            point(-b, 0., -a)
        };

        for (int i = 0; i < 20; ++i) {
            v[i].x = v[i].x * dodecahedron.radius / sqrt(3.) + dodecahedron.center.x;
            v[i].y = v[i].y * dodecahedron.radius / sqrt(3.) + dodecahedron.center.y;
            v[i].z = v[i].z * dodecahedron.radius / sqrt(3.) + dodecahedron.center.z;
        }

        polygons[6] = polygon(v[4], v[0], v[6], dodecahedron.color);
        polygons[7] = polygon(v[0], v[5], v[6], dodecahedron.color);
        polygons[8] = polygon(v[0], v[4], v[1], dodecahedron.color);
        polygons[9] = polygon(v[0], v[3], v[7], dodecahedron.color);
        polygons[10] = polygon(v[2], v[0], v[7], dodecahedron.color);
        polygons[11] = polygon(v[0], v[1], v[3], dodecahedron.color);
        polygons[12] = polygon(v[10], v[1], v[11], dodecahedron.color);
        polygons[13] = polygon(v[3], v[1], v[10], dodecahedron.color);
        polygons[14] = polygon(v[1], v[4], v[11], dodecahedron.color);
        polygons[15] = polygon(v[5], v[0], v[8], dodecahedron.color);
        polygons[16] = polygon(v[0], v[2], v[9], dodecahedron.color);
        polygons[17] = polygon(v[8], v[0], v[9], dodecahedron.color);
        polygons[18] = polygon(v[5], v[8], v[16], dodecahedron.color);
        polygons[19] = polygon(v[6], v[5], v[12], dodecahedron.color);
        polygons[20] = polygon(v[12], v[5], v[16], dodecahedron.color);
        polygons[21] = polygon(v[4], v[12], v[15], dodecahedron.color);
        polygons[22] = polygon(v[4], v[6], v[12], dodecahedron.color);
        polygons[23] = polygon(v[11], v[4], v[15], dodecahedron.color);
        polygons[24] = polygon(v[2], v[13], v[17], dodecahedron.color);
        polygons[25] = polygon(v[2], v[7], v[13], dodecahedron.color);
        polygons[26] = polygon(v[9], v[2], v[17], dodecahedron.color);
        polygons[27] = polygon(v[13], v[3], v[14], dodecahedron.color);
        polygons[28] = polygon(v[7], v[3], v[13], dodecahedron.color);
        polygons[29] = polygon(v[3], v[10], v[14], dodecahedron.color);
        polygons[30] = polygon(v[8], v[17], v[19], dodecahedron.color);
        polygons[31] = polygon(v[16], v[8], v[19], dodecahedron.color);
        polygons[32] = polygon(v[8], v[9], v[17], dodecahedron.color);
        polygons[33] = polygon(v[14], v[11], v[18], dodecahedron.color);
        polygons[34] = polygon(v[11], v[15], v[18], dodecahedron.color);
        polygons[35] = polygon(v[10], v[11], v[14], dodecahedron.color);
        polygons[36] = polygon(v[12], v[19], v[18], dodecahedron.color);
        polygons[37] = polygon(v[15], v[12], v[18], dodecahedron.color);
        polygons[38] = polygon(v[12], v[16], v[19], dodecahedron.color);
        polygons[39] = polygon(v[19], v[13], v[18], dodecahedron.color);
        polygons[40] = polygon(v[17], v[13], v[19], dodecahedron.color);
        polygons[41] = polygon(v[13], v[14], v[18], dodecahedron.color);
    }

    void init_icosahedron(polygon* polygons) {
        double a = icosahedron.radius / 0.951;
        double r = a * sin(36. * M_PI / 180.);
        double h = 0.25 * a * sqrt(3.);

        point v[] = {
            point(icosahedron.center.x, icosahedron.center.y, icosahedron.center.z),
            point(icosahedron.center.x, icosahedron.center.y + icosahedron.radius, icosahedron.center.z),
            point(icosahedron.center.x + r, icosahedron.center.y + h, icosahedron.center.z),
            point(icosahedron.center.x + r*sin(18. * M_PI/180.), icosahedron.center.y + h, icosahedron.center.z - r*cos(18. * M_PI/180.)),
            point(icosahedron.center.x + r*sin(18. * M_PI/180.), icosahedron.center.y + h, icosahedron.center.z + r*cos(18. * M_PI/180.)),
            point(icosahedron.center.x - r*sin(54. * M_PI/180.), icosahedron.center.y + h, icosahedron.center.z - r*cos(54. * M_PI/180.)),
            point(icosahedron.center.x - r*sin(54. * M_PI/180.), icosahedron.center.y + h, icosahedron.center.z + r*cos(54. * M_PI/180.)),
            point(icosahedron.center.x, icosahedron.center.y - icosahedron.radius, icosahedron.center.z),
            point(icosahedron.center.x - r, icosahedron.center.y - h, icosahedron.center.z),
            point(icosahedron.center.x - r*sin(18. * M_PI/180.), icosahedron.center.y - h, icosahedron.center.z - r*cos(18. * M_PI/180.)),
            point(icosahedron.center.x - r*sin(18. * M_PI/180.), icosahedron.center.y - h, icosahedron.center.z + r*cos(18. * M_PI/180.)),
            point(icosahedron.center.x + r*sin(54. * M_PI/180.), icosahedron.center.y - h, icosahedron.center.z - r*cos(54. * M_PI/180.)),
            point(icosahedron.center.x + r*sin(54. * M_PI/180.), icosahedron.center.y - h, icosahedron.center.z + r*cos(54. * M_PI/180.)),
        };

        polygons[42] = polygon(v[7], v[9], v[8], icosahedron.color);
        polygons[43] = polygon(v[7], v[8], v[10], icosahedron.color);
        polygons[44] = polygon(v[7], v[10], v[12], icosahedron.color);
        polygons[45] = polygon(v[7], v[12], v[11], icosahedron.color);
        polygons[46] = polygon(v[7], v[11], v[9], icosahedron.color);
        polygons[47] = polygon(v[8], v[5], v[6], icosahedron.color);
        polygons[48] = polygon(v[8], v[10], v[6], icosahedron.color);
        polygons[49] = polygon(v[10], v[12], v[4], icosahedron.color);
        polygons[50] = polygon(v[12], v[11], v[2], icosahedron.color);
        polygons[51] = polygon(v[11], v[9], v[3], icosahedron.color);
        polygons[52] = polygon(v[9], v[8], v[5], icosahedron.color);
        polygons[53] = polygon(v[10], v[6], v[4], icosahedron.color);
        polygons[54] = polygon(v[12], v[2], v[4], icosahedron.color);
        polygons[55] = polygon(v[11], v[3], v[2], icosahedron.color);
        polygons[56] = polygon(v[9], v[3], v[5], icosahedron.color);
        polygons[57] = polygon(v[3], v[2], v[1], icosahedron.color);
        polygons[58] = polygon(v[2], v[4], v[1], icosahedron.color);
        polygons[59] = polygon(v[4], v[6], v[1], icosahedron.color);
        polygons[60] = polygon(v[6], v[5], v[1], icosahedron.color);
        polygons[61] = polygon(v[5], v[3], v[1], icosahedron.color);
    }

public:
    app(bool _use_gpu) {
        use_gpu = _use_gpu;
        frames = frames_params{1, "%d.data", 800, 800, 90.};
        camera = camera_params{7., 3., 0., 2., 1., 2., 6., 1., 0., 0., 2., 0., 0., 0.5, 0.1, 1., 4., 1., 0., 0.};
        tetrahedron = figure_params{point(0., -2., 0.), make_uchar4(255, 0, 0, 255), 1.};
        dodecahedron = figure_params{point(0., 0., 0.), make_uchar4(0, 255, 0, 255), 1.};
        icosahedron = figure_params{point(0., 2., 0.), make_uchar4(0, 0, 255, 255), 1.};
        floor = floor_params{point(-5., -5., -1.), point(-5., 5., -1.), point(5., 5., -1.), point(5., -5., -1), make_uchar4(255, 255, 255, 255)};
        light = light_params{point(10., 0., 15.), make_uchar4(75, 50, 25, 255), 4.};
    }

    app(
        bool _use_gpu,
        frames_params _frames, 
        camera_params _camera, 
        figure_params _tetrahedron,
        figure_params _dodecahedron,
        figure_params _icosahedron,
        floor_params _floor,
        light_params _light
    ) {
        use_gpu = _use_gpu;
        frames = _frames;
        camera = _camera;
        tetrahedron = _tetrahedron;
        dodecahedron = _dodecahedron;
        icosahedron = _icosahedron;
        floor = _floor;
        light = _light;
    }

    void run() {
        uchar4* data = (uchar4*)malloc(sizeof(uchar4) * frames.width * frames.height * light.sqrt_rpp * light.sqrt_rpp);
        uchar4* smoothing_data = (uchar4*)malloc(sizeof(uchar4) * frames.width * frames.height);
        uchar4* dev_data;
        uchar4* dev_smoothing_data;
        polygon polygons[polygons_amount];
        polygon* dev_polygons;

        init_floor(polygons);
        init_tetrahedron(polygons);
        init_dodecahedron(polygons);
        init_icosahedron(polygons);
        
        if (use_gpu) {
            hipMalloc(&dev_data, sizeof(uchar4) * frames.width * frames.height * light.sqrt_rpp * light.sqrt_rpp);
            hipMalloc(&dev_smoothing_data, sizeof(uchar4) * frames.width * frames.height);
            hipMalloc(&dev_polygons, sizeof(polygon) * polygons_amount);
            hipMemcpy(dev_polygons, polygons, sizeof(polygon) * polygons_amount, hipMemcpyHostToDevice);
        }

        for (int i = 0; i < frames.amount; ++i) {
            double t = 2 * M_PI * i / frames.amount;
            point camera_current_position = point(
                (camera.r0c + camera.arc * sin(camera.wrc * t + camera.prc)) * cos(camera.phi0c + camera.wphic * t),
                (camera.r0c + camera.arc * sin(camera.wrc * t + camera.prc)) * sin(camera.phi0c + camera.wphic * t),
                camera.z0c + camera.azc * sin(camera.wzc * t + camera.pzc)
            );
            point camera_current_view = point(
                (camera.r0n + camera.arn * sin(camera.wrn * t + camera.prn)) * cos(camera.phi0n + camera.wphin * t),
                (camera.r0n + camera.arn * sin(camera.wrn * t + camera.prn)) * sin(camera.phi0n + camera.wphin * t),
                camera.z0n + camera.azn * sin(camera.wzn * t + camera.pzn)
            );

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);

            if (use_gpu) {
                gpu_render<<<dim3(x_blocks_amount, x_threads_amount), dim3(y_blocks_amount, y_threads_amount)>>>(
                    dev_data,
                    camera_current_position, 
                    camera_current_view, 
                    frames.width * light.sqrt_rpp, 
                    frames.height * light.sqrt_rpp, 
                    frames.view_angle,
                    light.position, 
                    light.color, 
                    dev_polygons, 
                    polygons_amount
                );

                gpu_smoothing<<<dim3(x_blocks_amount, x_threads_amount), dim3(y_blocks_amount, y_threads_amount)>>>(
                    dev_data, 
                    dev_smoothing_data, 
                    frames.width, 
                    frames.height, 
                    light.sqrt_rpp
                );

                hipMemcpy(smoothing_data, dev_smoothing_data, sizeof(uchar4) * frames.width * frames.height, hipMemcpyDeviceToHost);
            } else {
                cpu_render(
                    data,
                    camera_current_position, 
                    camera_current_view, 
                    frames.width * light.sqrt_rpp, 
                    frames.height * light.sqrt_rpp, 
                    frames.view_angle,
                    light.position, 
                    light.color, 
                    polygons, 
                    polygons_amount
                );

                cpu_smoothing(
                    data, 
                    smoothing_data, 
                    frames.width, 
                    frames.height, 
                    light.sqrt_rpp
                );
            }

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);

            
            char path_to_save_ith_frame[frames.path_to_save_frames.length() + std::to_string(i).length() - 2];
            sprintf(path_to_save_ith_frame, frames.path_to_save_frames.c_str(), i);
            FILE* output_file = fopen(path_to_save_ith_frame, "w");
            fwrite(&frames.width, sizeof(int), 1, output_file);
            fwrite(&frames.height, sizeof(int), 1, output_file);
            fwrite(smoothing_data, sizeof(uchar4), frames.width * frames.height, output_file);
            fclose(output_file);
        }

        free(data);
        free(smoothing_data);
        if (use_gpu) {
            hipFree(dev_data);
            hipFree(dev_smoothing_data);
            hipFree(dev_polygons);
        }
    }

    ~app() {}

    friend std::ostream& operator<<(std::ostream& out, app& a) {
        out << a.frames.amount << std::endl;
        out << a.frames.path_to_save_frames << std::endl;
        out << a.frames.width << " " << a.frames.height << " " << a.frames.view_angle << std::endl;
        
        out << a.camera.r0c << " " << a.camera.z0c << " " << a.camera.phi0c << " " 
            << a.camera.arc << " " << a.camera.azc << " " << a.camera.wrc << " " 
            << a.camera.wzc << " " << a.camera.wphic << " " << a.camera.prc << " " 
            << a.camera.pzc << std::endl;
        out << a.camera.r0n << " " << a.camera.z0n << " " << a.camera.phi0n << " " 
            << a.camera.arn << " " << a.camera.azn << " " << a.camera.wrn << " " 
            << a.camera.wzn << " " << a.camera.wphin << " " << a.camera.prn << " " 
            << a.camera.pzn << std::endl;

        out << a.tetrahedron.center << " " << int(a.tetrahedron.color.x) / 255. << " " 
            << int(a.tetrahedron.color.y) / 255. << " " << int(a.tetrahedron.color.z) / 255. << " " 
            << a.tetrahedron.radius << std::endl;

        out << a.dodecahedron.center << " " << int(a.dodecahedron.color.x) / 255. << " " 
            << int(a.dodecahedron.color.y) / 255. << " " << int(a.dodecahedron.color.z) / 255. << " " 
            << a.dodecahedron.radius << std::endl;

        out << a.icosahedron.center << " " << int(a.icosahedron.color.x) / 255. << " " 
            << int(a.icosahedron.color.y) / 255. << " " << int(a.icosahedron.color.z) / 255. << " " 
            << a.icosahedron.radius << std::endl;

        out << a.floor.p1 << " " << a.floor.p2 << " " << a.floor.p3 << " " << a.floor.p4 << " " 
            << int(a.floor.color.x) / 255. << " " << int(a.floor.color.y) / 255. << " " << int(a.floor.color.z) / 255. << std::endl;

        out << a.light.position << " " << int(a.light.color.x) / 255. << " " << int(a.light.color.y) / 255. << " "
            << int(a.light.color.z) / 255. << " " << a.light.sqrt_rpp;
        
        return out;
    }
};

int main(int args, char** argv) {
    bool gpu_flag = false, cpu_flag = false, default_flag = false;
    for (int i = 1; i < args; ++i) {
        std::string argument = std::string(argv[i]);
        if (argument == "--cpu") {
            cpu_flag = true;
        } else if (argument == "--gpu") {
            gpu_flag = true;
        } else if (argument == "--default") {
            default_flag = true;
        }
    }
    bool use_gpu = true;
    if (cpu_flag && !gpu_flag) {
        use_gpu = false;
    }

    /* std::chrono::steady_clock::time_point start = 
        std::chrono::steady_clock::now(); */

    if (default_flag) {
        app App = app(use_gpu);
        std::cout << App << std::endl;
        App.run();
    } else {
        frames_params frames;
        camera_params camera;
        figure_params tetrahedron;
        figure_params dodecahedron;
        figure_params icosahedron;
        floor_params floor;
        light_params light;

        std::cin >> frames;
        std::cin >> camera;
        std::cin >> tetrahedron;
        std::cin >> dodecahedron;
        std::cin >> icosahedron;
        std::cin >> floor;
        std::cin >> light;

        app App = app(
            use_gpu,
            frames,
            camera,
            tetrahedron,
            dodecahedron,
            icosahedron,
            floor,
            light
        );
        App.run();
    }

    /* std::chrono::steady_clock::time_point finish = 
        std::chrono::steady_clock::now();
    unsigned time = 
        std::chrono::duration_cast<std::chrono::seconds>(finish - start).count();

    printf("time: %d sec\n", time); */

    return 0;
}
