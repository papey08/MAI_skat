
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <chrono>

const int THREADS_AMOUNT = 32;
const int BLOCKS_AMOUNT = 32;

__device__ double get_min(double d1, double d2) {
    if (d1 < d2) {
        return d1;
    } else {
        return d2;
    }
}

__global__ void kernel(double *arr1, double *arr2, double *res, int size) {
    int offset = gridDim.x * blockDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += offset) {
        res[i] = get_min(arr1[i], arr2[i]);
    }
}

int main() {
    int n;
    scanf("%d", &n);

    double *arr1 = (double*)malloc(sizeof(double) * n);
    for (int i = 0; i < n; ++i) {
        scanf("%lf", &arr1[i]);
    }

    double *arr2 = (double*)malloc(sizeof(double) * n);
    for (int i = 0; i < n; ++i) {
        scanf("%lf", &arr2[i]);
    }

    double *dev_arr1;
    hipMalloc(&dev_arr1, sizeof(double) * n);
    hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice);

    double *dev_arr2;
    hipMalloc(&dev_arr2, sizeof(double) * n);
    hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    double *dev_res;
    hipMalloc(&dev_res, sizeof(double) * n);

    /* std::chrono::steady_clock::time_point start = 
        std::chrono::steady_clock::now(); */

    kernel<<<BLOCKS_AMOUNT, THREADS_AMOUNT>>>(dev_arr1, dev_arr2, dev_res, n);

    /* std::chrono::steady_clock::time_point finish = 
        std::chrono::steady_clock::now();
    unsigned time = 
        std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count(); */

    double *res = (double*)malloc(sizeof(double) * n);
    hipMemcpy(res, dev_res, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        printf("%.10lf ", res[i]);
    }
    printf("\n");
    // printf("time: %dns\n", time);

    free(arr1);
    free(arr2);
    free(res);
    hipFree(dev_arr1);
    hipFree(dev_arr2);
    hipFree(dev_res);

    return 0;
}
