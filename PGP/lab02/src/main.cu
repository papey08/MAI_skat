#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include "math.h"
// #include <chrono>

const int X_BLOCKS = 8;
const int X_THREADS = 8;
const int Y_BLOCKS = 8;
const int Y_THREADS = 8;

class image{
public:
    int width;
    int height;

    uchar4* pixels;

    image(std::string path) {
        FILE *img = fopen(path.c_str(), "rb");
        fread(&width, sizeof(int), 1, img);
        fread(&height, sizeof(int), 1, img);
        pixels = (uchar4*)malloc(sizeof(uchar4) * width * height);
        fread(pixels, sizeof(uchar4), width * height, img);
        fclose(img);
    }

    ~image() {
        free(pixels);
    }

    void save_to_file(std::string path) {
        FILE *img = fopen(path.c_str(), "wb");
        fwrite(&width, sizeof(int), 1, img);
        fwrite(&height, sizeof(int), 1, img);
        fwrite(pixels, sizeof(uchar4), width * height, img);
        fclose(img);
    }
};

const double R_COEFF = 0.299;
const double G_COEFF = 0.587;
const double B_COEFF = 0.114;

__device__ double to_black_white(uchar4 p) {
    return R_COEFF * p.x + G_COEFF * p.y + B_COEFF * p.z;
}

__device__ uchar4 sobel(double w[3][3]) {
    double gx = w[0][2] + 2 * w[1][2] + w[2][2] - w[0][0] - 2 * w[1][0] - w[2][0];
    double gy = w[2][0] + 2 * w[2][1] + w[2][2] - w[0][0] - 2 * w[0][1] - w[0][2];

    int res = min(255, int(sqrt(gx * gx + gy * gy)));
    return make_uchar4(res, res, res, res);
}

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int width, int height) {
    int i_y = blockDim.y * blockIdx.y + threadIdx.y;
    int i_x = blockDim.x * blockIdx.x + threadIdx.x;
    int offset_y = blockDim.y * gridDim.y;
    int offset_x = blockDim.x * gridDim.x;

    for (int y = i_y; y < height; y += offset_y) {
        for (int x = i_x; x < width; x += offset_x) {
            double w[3][3];

            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    w[i][j] = to_black_white(tex2D(tex, x-1+i, y-1+j));
                }
            }

            out[y*width + x] = sobel(w);
        }
    }
}

int main() {
    std::string in_path, out_path;
    std::cin >> in_path >> out_path;
    image img(in_path);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    hipMallocArray(&arr, &ch, img.width, img.height);
    hipMemcpyToArray(arr, 0, 0, img.pixels, sizeof(uchar4) * img.width * img.height, hipMemcpyHostToDevice);

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    hipBindTextureToArray(tex, arr, ch);
    uchar4* dev_out;
    hipMalloc(&dev_out, sizeof(uchar4) * img.width * img.height);

    /* std::chrono::steady_clock::time_point start = 
        std::chrono::steady_clock::now(); */

    kernel<<<dim3(X_BLOCKS, X_THREADS), dim3(Y_BLOCKS, Y_THREADS)>>>(dev_out, img.width, img.height);

    /* std::chrono::steady_clock::time_point finish = 
        std::chrono::steady_clock::now();
    unsigned time = 
        std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();

    printf("time: %dns\n", time); */

    hipMemcpy(img.pixels, dev_out, sizeof(uchar4) * img.width * img.height, hipMemcpyDeviceToHost);
    
    hipUnbindTexture(tex);
    hipFreeArray(arr);
    hipFree(dev_out);

    img.save_to_file(out_path);

    return 0;
}
