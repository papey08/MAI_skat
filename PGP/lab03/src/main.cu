#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <math.h>

const int X_BLOCKS = 8;
const int X_THREADS = 8;
const int Y_BLOCKS = 8;
const int Y_THREADS = 8;

__constant__ double average_cache[32][3];
__constant__ double normalized_cache[32][3];
double average_buffer[32][3];
double normalized_buffer[32][3];

class image {
public:
    int width;
    int height;

    uchar4* pixels;

    image(std::string path) {
        FILE *img = fopen(path.c_str(), "rb");
        fread(&width, sizeof(int), 1, img);
        fread(&height, sizeof(int), 1, img);
        pixels = (uchar4*)malloc(sizeof(uchar4) * width * height);
        fread(pixels, sizeof(uchar4), width * height, img);
        fclose(img);
    }

    ~image() {
        free(pixels);
    }

    void save_to_file(std::string path) {
        FILE *img = fopen(path.c_str(), "wb");
        fwrite(&width, sizeof(int), 1, img);
        fwrite(&height, sizeof(int), 1, img);
        fwrite(pixels, sizeof(uchar4), width * height, img);
        fclose(img);
    }
};

void fill_average_buffer(std::vector<std::vector<std::pair<int, int>>>& coords, uchar4* pixels, int width, int height, int classes_amount) {
    std::vector<double> avgs(32*3);
    for (int i = 0; i < classes_amount; i++) {
        avgs[i*3+0] = 0;
        avgs[i*3+1] = 0;
        avgs[i*3+2] = 0;
        for (int j = 0; j < coords[i].size(); j++) {
            uchar4 pixel = pixels[coords[i][j].second * width + coords[i][j].first];
            double rgb[3];
			rgb[0] = pixel.x;
    		rgb[1] = pixel.y;
    		rgb[2] = pixel.z;
            avgs[i*3+0] += rgb[0];
            avgs[i*3+1] += rgb[1];
            avgs[i*3+2] += rgb[2];
        }
        avgs[i*3+0] /= coords[i].size();
        avgs[i*3+1] /= coords[i].size();
        avgs[i*3+2] /= coords[i].size();
    }
    for (int i = 0; i < classes_amount; i++) {
        average_buffer[i][0] = avgs[i*3+0];
        average_buffer[i][1] = avgs[i*3+1];
        average_buffer[i][2] = avgs[i*3+2];
    }
}

__device__ double spectral_angle_method(uchar4 pixel, int avg) {
    double rgb_pixel[3];
    rgb_pixel[0] = pixel.x;
    rgb_pixel[1] = pixel.y;
    rgb_pixel[2] = pixel.z;
    double t_rgb[3];
    double t_normalized[3];
    t_rgb[0] = rgb_pixel[0];
	t_rgb[1] = rgb_pixel[1];
	t_rgb[2] = rgb_pixel[2];
    t_normalized[0] = normalized_cache[avg][0];
	t_normalized[1] = normalized_cache[avg][1];
	t_normalized[2] = normalized_cache[avg][2];
    return t_rgb[0] * t_normalized[0] + t_rgb[1] * t_normalized[1] + t_rgb[2] * t_normalized[2];
}

__global__ void kernel(uchar4* pixels, int width, int height, int classes_amount) {
    int i_x = blockDim.x * blockIdx.x + threadIdx.x;
    int i_y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    for (int y = i_y; y < height; y += offset_y) {
        for (int x = i_x; x < width; x += offset_x) {
            uchar4 pixel = pixels[y * width + x];
            double coeff1 = spectral_angle_method(pixel, 0);
            int idx = 0;
            for (int i = 1; i < classes_amount; i++) {
                double argmax = spectral_angle_method(pixel, i);
                if (coeff1 < argmax) {
                    coeff1 = argmax;
                    idx = i;
                }
            }
            pixels[y*width + x].w = (unsigned char)idx;
        }
	}
}
int main() {
    std::string in_path, out_path;
    std::cin >> in_path >> out_path;
    image img(in_path);
    int classes_amount;
    std::cin >> classes_amount;
    std::vector<std::vector<std::pair<int, int>>> coords(classes_amount);
    for (int i = 0; i < classes_amount; i++) {
		int pairs_amount;
        std::cin >> pairs_amount;
        coords[i].resize(pairs_amount);
        for (int j = 0; j < pairs_amount; j++) {
            std::cin >> coords[i][j].first >> coords[i][j].second;
        }
    }
    fill_average_buffer(coords, img.pixels, img.width, img.height, classes_amount);
    for (int i = 0; i < classes_amount; i++) {
        normalized_buffer[i][0] = (double)average_buffer[i][0] / sqrt(pow(average_buffer[i][0], 2) + pow(average_buffer[i][1], 2) + pow(average_buffer[i][2], 2));
        normalized_buffer[i][1] = (double)average_buffer[i][1] / sqrt(pow(average_buffer[i][0], 2) + pow(average_buffer[i][1], 2) + pow(average_buffer[i][2], 2));
        normalized_buffer[i][2] = (double)average_buffer[i][2] / sqrt(pow(average_buffer[i][0], 2) + pow(average_buffer[i][1], 2) + pow(average_buffer[i][2], 2));
    }
    hipMemcpyToSymbol(HIP_SYMBOL(average_cache), average_buffer, 32 * sizeof(double[3]));
    hipMemcpyToSymbol(HIP_SYMBOL(normalized_cache), normalized_buffer, 32 * sizeof(double[3]));
    uchar4* dev_out;
    hipMalloc(&dev_out, sizeof(uchar4) * img.width * img.height);
    hipMemcpy(dev_out, img.pixels, sizeof(uchar4) * img.width * img.height, hipMemcpyHostToDevice);
    kernel<<<dim3(X_BLOCKS, X_THREADS), dim3(Y_BLOCKS, Y_THREADS)>>>(dev_out, img.width, img.height, classes_amount);
    hipGetLastError();
    hipMemcpy(img.pixels, dev_out, sizeof(uchar4) * img.width * img.height, hipMemcpyDeviceToHost);
    hipFree(dev_out);
    img.save_to_file(out_path);
    return 0;
}
		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 
                                                                                                                                                 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 				 