#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

const int X_BLOCKS = 32;
const int X_THREADS = 32;
const int Y_BLOCKS = 32;
const int Y_THREADS = 32;
const int BLOCKS = 1024;
const int THREADS = 1024;

const double EPS = 10e-7;

__global__ void div_row(double* dev_data, int shape, double delim, int r) {
    int offset = blockDim.x * gridDim.x;
    for (int i = r + blockDim.x*blockIdx.x + threadIdx.x; i < shape; i += offset) {
        dev_data[r + i*shape] /= delim;
    }
}

__global__ void swap_rows(double* dev_data, int shape, int r1, int r2) {
    int offset = blockDim.x * gridDim.x;
    for (int i = r1 + blockDim.x * blockIdx.x + threadIdx.x; i < shape; i += offset) {
        double temp = dev_data[i*shape + r1];
        dev_data[i*shape + r1] = dev_data[i*shape + r2];
        dev_data[i*shape + r2] = temp;
    }
}

__global__ void kernel(double* dev_data, int shape, int r) {
    int x_offset = blockDim.x * gridDim.x;
    int y_offset = blockDim.y * gridDim.y;
    for (int i = r + blockDim.x*blockIdx.x + threadIdx.x + 1; i < shape; i += x_offset) {
        for (int j = r + blockDim.y*blockIdx.y + threadIdx.y + 1; j < shape; j += y_offset) {
            dev_data[j*shape + i] -= dev_data[r*shape + i] * dev_data[r + j*shape];
        }
    }
}

struct comparator {
    __host__ __device__ bool operator()(double a, double b) {
        return std::fabs(a) < std::fabs(b);
    }
};

class Matrix {
public:
    Matrix(int n) : shape(n) {
        data = (double*)malloc(sizeof(double) * n * n);
    }

    double determinant() {
        comparator comp;                                                                                                                                                    
        double* dev_matrix;                                                                                                                                                    
        hipMalloc(&dev_matrix, sizeof(double) * shape * shape);                                                                                                                                                    
        hipMemcpy(dev_matrix, data, sizeof(double) * shape * shape, hipMemcpyHostToDevice);                                                                                                                                                    
        std::vector<double> delims(shape, 0.);                                                                                                                                                    
        thrust::device_ptr<double> left_ptr;                                                                                                                                                    
        thrust::device_ptr<double> max_ptr;                                                                                                                                                    
        for (int i = 0; i < shape; ++i) {                                                                                                                                                    
            left_ptr = thrust::device_pointer_cast(dev_matrix + i + i * shape);                                                                                                                                                    
            double delim = *left_ptr;                                                                                                                                                    
            if (std::abs(delim) <= EPS) {                                                                                                                                                    
                max_ptr = thrust::max_element(left_ptr, left_ptr + (shape - i), comp);                                                                                                                                                    
                double max_elem = *max_ptr;                                                                                                                                                    
                int max_index = max_ptr - left_ptr;                                                                                                                                                    
                if (std::abs(max_elem) <= EPS) {                                                                                                                                                    
                    hipFree(dev_matrix);                                                                                                                                                    
                    return 0.;                                                                                                                                                    
                }                                                                                                                                                    
                swap_rows<<<BLOCKS, THREADS>>>(dev_matrix, shape, i, max_index+i);                                                                                                                                                    
                delims[i] -= max_elem;                                                                                                                                                    
                delim = max_elem;                                                                                                                                                    
            }                                                                                                                                                    
            else {                                                                                                                                                    
                delims[i] = delim;                                                                                                                                                    
            }                                                                                                                                                    
            div_row<<<BLOCKS, THREADS>>>(dev_matrix, shape, delim, i);                                                                                                                                                    
            kernel<<<dim3(X_BLOCKS, X_THREADS), dim3(Y_BLOCKS, Y_THREADS)>>>(dev_matrix, shape, i);                                                                                                                                                    
        }                                                                                                                                                    
        hipFree(dev_matrix);                                                                                                                                                    
        std::sort(delims.begin(), delims.end(), comp);                                                                                                                                                    
        double det = 1.0;                                                                                                                                                    
        int l = 0;                                                                                                                                                    
        int r = delims.size() - 1;                                                                                                                                                    
        while (l <= r) {                                                                                                                                                    
            if (std::abs(det) < EPS) {                                                                                                                                                    
                det *= delims[r];                                                                                                                                                    
                --r;                                                                                                                                                    
            } else {                                                                                                                                                    
                det *= delims[l];                                                                                                                                                    
                ++l;                                                                                                                                                    
            }                                                                                                                                                    
        }                                                                                                                                                    
        return det;                                                                                                                                                    
    }

    friend std::istream& operator>>(std::istream& in, Matrix& matrix) {
        for (int i = 0; i < matrix.shape; ++i) {
            for (int j = 0; j < matrix.shape; ++j) {
                in >> matrix.data[i + j*matrix.shape];
            }
        }
        return in;
    }

    ~Matrix() {
        free(data);
    }

private:
    double* data;
    int shape;
};

int main() {
    std::ios::sync_with_stdio(false);
    std::cin.tie(nullptr);
    std::cout.tie(nullptr);
    int n;
    std::cin >> n;
    Matrix m = Matrix(n);
    std::cin >> m;
    std::cout << m.determinant() << std::endl;
    return 0;
}
